#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

#include "aux.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


int *device_alloc_int(int size){
    int *out;
    gpuErrchk(hipMalloc(&out, sizeof(int) * size));
    return out;
}

float *device_alloc_float(int size){
    float *out;
    gpuErrchk(hipMalloc(&out, sizeof(float) * size));
    return out;
}

void copy_to_device_int(int *host_ptr, int *device_ptr, int count){
    gpuErrchk(hipMemcpy(device_ptr, host_ptr, sizeof(int) * count, hipMemcpyHostToDevice));
}

void copy_from_device_int(int *host_ptr, int *device_ptr, int count){
    gpuErrchk(hipMemcpy(host_ptr, device_ptr, sizeof(int) * count, hipMemcpyDeviceToHost));
}

void copy_to_device_float(float *host_ptr, float *device_ptr, int count){
    gpuErrchk(hipMemcpy(device_ptr, host_ptr, sizeof(float) * count, hipMemcpyHostToDevice));
}

void copy_from_device_float(float *host_ptr, float *device_ptr, int count){
    gpuErrchk(hipMemcpy(host_ptr, device_ptr, sizeof(float) * count, hipMemcpyDeviceToHost));
}

void device_free(void *ptr){
    gpuErrchk(hipFree(ptr));
}